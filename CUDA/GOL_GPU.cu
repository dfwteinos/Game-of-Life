#include "hip/hip_runtime.h"
#include "GOL_CUDA.h"
#include <hip/hip_runtime_api.h>
#include "lcutil.h"
#include "timestamp.h"

__global__ void GOL_Simulation(int rows, int columns, int plainSize, char *prevPlain, char *currPlain)
{
    const unsigned int thread_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int totalNeighbours, upperNeighbours, sideNeighbours, lowerNeighbours;
    //total number of pixels up to the row over the current point
    //and the number of pixels up to the current row
    //simplifying the expressions for neighbouring positions
    int upperPlainPixels, pixelsUpToRow;
    //the y coordinates on the left and right of the current point
    //keeping in mind our plain is interconnected
    int leftY, rightY;
    //coordinates of current point
    int currX, currY;

    //taking upper bound number of blocks
    //some threads will be obsolete
    if(thread_pos < plainSize)
    {
        currX = thread_pos / columns;
        currY = thread_pos - currX * columns;

        leftY = (currY + columns - 1) % columns;
        rightY = (currY + 1) % columns;

        upperPlainPixels = columns * ((currX + rows - 1) % rows);
        pixelsUpToRow = columns * ((currX + 1) % rows);

        //calculating the neighbours
        upperNeighbours = prevPlain[upperPlainPixels + leftY] + prevPlain[upperPlainPixels + currY] + prevPlain[upperPlainPixels + rightY];
        sideNeighbours = prevPlain[currX * columns + leftY] + prevPlain[currX * columns + rightY];
        lowerNeighbours = prevPlain[pixelsUpToRow + leftY] + prevPlain[pixelsUpToRow + currY] + prevPlain[pixelsUpToRow + rightY];
        //add the three groups of neighbours
        totalNeighbours = upperNeighbours + sideNeighbours + lowerNeighbours;

        //pixel survives
        if(totalNeighbours == 3 || (totalNeighbours == 2 && prevPlain[thread_pos] == 1))
        {
            currPlain[thread_pos] = 1;
        }else//pixel dies
        {
            currPlain[thread_pos] = 0;
        }
    }
}


//function used to print error messages for cuda errors
int checkFunction(hipError_t functionOutput, const char *msg)
{
    if (functionOutput != hipSuccess)
    {
        perror(msg);
        exit(functionOutput);
    }
    return 0;
}


extern "C" float GPU_GOL(int rows, int columns, int iterations, char **prevPlain, char **currPlain)
{
    int totalPixels = rows * columns;
    int plainMemory = totalPixels * sizeof(char);
    int iteration;
    double microseconds;
    //plains stored in gpu memory
    char *d_prevPlain, *d_currPlain, *d_tmpPlain;
    //block and threads variables for CUDA code
    const int THREADS_PER_BLOCK = 1024;
    dim3 blockNum(THREADS_PER_BLOCK);
    dim3 threadNum(FRACTION_CEILING(totalPixels, THREADS_PER_BLOCK));

    //allocating memory for the GPU plains and copying data from CPU to GPU
    checkFunction(hipMalloc((void**) &d_prevPlain, plainMemory), "Failed to allocate memory for GPU previous plain");
    checkFunction(hipMemcpy(d_prevPlain, &(prevPlain[0][0]), plainMemory, hipMemcpyHostToDevice), "Failed to copy plain from CPU to GPU");
    checkFunction(hipMalloc((void**) &d_currPlain, plainMemory), "Failed to allocate memory for GPU previous plain");

    timestamp t_start;
    t_start = getTimestamp();
    //actual simulation execution
    for(iteration = 0; iteration < iterations; iteration++)
    {
        GOL_Simulation<<<threadNum, blockNum>>>(rows, columns, totalPixels, d_prevPlain, d_currPlain);
        //checking whether an error occured during current GOL simulation's iteration
        checkFunction(hipGetLastError(), "Error occured during GPU execution");
        //current plain is the previous updated one
        d_tmpPlain = d_prevPlain;
        d_prevPlain = d_currPlain;
        d_currPlain = d_tmpPlain;
    }
    hipDeviceSynchronize();
    microseconds = getElapsedtime(t_start);
    //copy the final plain from gpu to cpu memory
    checkFunction(hipMemcpy(&(currPlain[0][0]), d_currPlain, plainMemory, hipMemcpyDeviceToHost), "Failed to copy plain from GPU to CPU");
    //freeing the memory allocated in the gpu
    checkFunction(hipFree(d_prevPlain), "Failed to free GPU previous plain");
    checkFunction(hipFree(d_currPlain), "Failed to free GPU current plain");
    
    return microseconds;
}